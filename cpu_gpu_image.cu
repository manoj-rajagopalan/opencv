#include "cpu_gpu_image.hpp"

#include "hip/hip_runtime.h"

namespace {

void check(hipError_t const cuda_status, std::string const& stage)
{
    if(hipSuccess != cuda_status) {
        std::cout << "CUDA ERROR " << cuda_status << " in " << stage << std::endl;
        std::cout << "- " << hipGetErrorName(cuda_status) << ": " << hipGetErrorString(cuda_status) << std::endl;
    }
}

}


CpuGpuImage::CpuGpuImage(int32_t const width, int32_t const height, std::string const& name)
: width_(width),
  height_(height),
  cpu_mat_(width, height, CV_8UC3),
  name_(name)
{
hipError_t cuda_status = hipSuccess;

cuda_status = hipMalloc((void**) &gpu_ptr_, width * height * 3);
check(cuda_status, "GPU buffer allocation " + name_);
}

CpuGpuImage::~CpuGpuImage()
{
    cuda_status = hipFree((void*) gpu_ptr_);
    check(cuda_status, "GPU buffer release" + name_);
}

void CpuGpuImage::copyToGpu()
{
    hipError_t cuda_status = hipMemcpy((void*) gpu_ptr_, (void const *) cpu_mat_.ptr(), width_ * height_ * 3, hipMemcpyHostToDevice);
    check(cuda_status, "CPU --> GPU copy" + name_);
}

void CpuGpuImage::copyFromGpu()
{
    hipError_t cuda_status = hipMemcpy((void*) cpu_mat_.ptr(), (void const *) gpu_ptr_, width_ * height_ * 3, hipMemcpyDeviceToHost);
    check(cuda_status, "GPU --> CPU copy" + name_);
}
