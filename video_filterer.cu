#include "hip/hip_runtime.h"
/// Processes input video file and writes to output

#include <cassert>
#include <chrono>
#include <fstream>
#include <iostream>
#include <sstream>

#include "hip/hip_runtime.h"

#include "opencv2/videoio.hpp"

struct VideoProperties
{
    int num_frames;
    cv::Size frame_size;
    double fps;
    bool is_color;
    int cv_mat_type;
};

VideoProperties properties(cv::VideoCapture const& video_infile)
{
    VideoProperties video_properties;
    video_properties.num_frames = int(video_infile.get(cv::CAP_PROP_FRAME_COUNT));
    video_properties.frame_size.width = int(video_infile.get(cv::CAP_PROP_FRAME_WIDTH));
    video_properties.frame_size.height = int(video_infile.get(cv::CAP_PROP_FRAME_HEIGHT));
    video_properties.fps = video_infile.get(cv::CAP_PROP_FPS);
    video_properties.cv_mat_type = int(video_infile.get(cv::CAP_PROP_FORMAT));
    video_properties.is_color = true;
    return video_properties;
}

__constant__ float filter_gpu[25];

__global__
void cudaFilter(uchar3 *const out_image,
                const uchar3 *const in_image,
                int const num_pixels,
                int const image_width,
                int const image_height,
                int const filter_width,
                int const filter_height)
{
    int const tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < num_pixels) {
        int const x = tid % image_width;
        int const y = tid / image_width;
        
        float sum_x = 0.0f, sum_y = 0.0f, sum_z = 0.0f;
        int const window_right = filter_width / 2;
        int const window_left = -window_right;
        int const window_bottom = filter_height / 2;
        int const window_top = -window_bottom;
        for(int i = window_top; i <= window_bottom; ++i) {
            int yi = y + i;
            yi = yi < 0 ? 0 : (yi >= image_height ? (image_height-1) : yi);

            for(int j = window_left; j <= window_right; ++j) {
                int xj = x + j;
                xj = xj < 0 ? 0 : (xj >= image_width ? (image_width-1) : xj);

                int const pixel_index = yi * image_width + xj;
                uchar3 const pixel = in_image[pixel_index];

                float const filter_coeff = filter_gpu[(i - window_top) * filter_width + (j - window_left)];
                sum_x += filter_coeff * pixel.x;
                sum_y += filter_coeff * pixel.y;
                sum_z += filter_coeff * pixel.z;
            }
        }
        uchar3 pixel = make_uchar3(int(sum_x), int(sum_y), int(sum_z));
        out_image[tid] = pixel;
    }

}

void cudaCheckSuccess(hipError_t const cuda_status, std::string const& message)
{
    if(hipSuccess != cuda_status) {
        std::cout << "CUDA ERROR " << cuda_status << ": " << message << std::endl;
        std::cout << "- " << hipGetErrorName(cuda_status) << ": " << hipGetErrorString(cuda_status) << std::endl;
    }
}

class Filter
{
public:
    Filter() {}
    ~Filter() {}

    int width() const { return width_; }
    int height() const { return height_; }
    int area() const { return int(data_.size()); }

    float const* data() const { return data_.data(); }

    void loadFromFile(std::string const& file_name) {
        std::ifstream file(file_name.c_str());
        assert(file && "Unable to open filter file");
        while(file.good()) {
            ++height_;
            std::string line;
            std::getline(file, line);
            std::istringstream line_input(line);
            int line_width = 0;
            float param = 0.0f;
            while(line_input) {
                param = std::nanf("");
                line_input >> param;
                if(std::isnan(param)) {
                    break;
                }
                data_.push_back(param);
                ++line_width;
            }
            assert(width_ == 0 || line_width == width_);
            width_ = line_width;
        }
        assert((width_ * height_) == int(data_.size()) && "Error parsing filter");
    }

    void print(std::ostream& o, std::string const& line_begin = "") const {
        auto data_iter = data_.cbegin();
        for(int i = 0; i < height_; ++i) {
            o << line_begin;
            for(int j = 0; j < width_; ++j) {
                o << *data_iter << ' ';
                ++data_iter;
            }
            o << '\n';
        }
    }

private:
    int width_ = 0;
    int height_ = 0;
    std::vector<float> data_;
};

int main(int const argc, char const *const argv[])
{
    assert(argc == 4 && "Usage: video_filterer <infile> <outfile> <filter file (row major order, max 25 elements)");

    cv::VideoCapture video_infile;
    bool status = video_infile.open(argv[1]);
    assert(status && "Unable to open input file");
    VideoProperties const video_properties = properties(video_infile);
    std::cout << "Input mat type = " << video_properties.cv_mat_type << std::endl;
    // assert(video_properties.cv_mat_type == CV_8UC3);

    cv::VideoWriter video_outfile;
    status = video_outfile.open(argv[2], cv::VideoWriter::fourcc('H','2','6','4'),
                                video_properties.fps, video_properties.frame_size, video_properties.is_color);

    Filter filter_cpu;
    filter_cpu.loadFromFile(argv[3]);
    std::cout << "Filter dims: " << filter_cpu.width() << 'x' << filter_cpu.height() << " = " << filter_cpu.area() << std::endl;
    filter_cpu.print(std::cout, "\t");
    hipMemcpyToSymbol(HIP_SYMBOL(filter_gpu), (const void*) filter_cpu.data(), filter_cpu.area(), /*offset=*/ 0, hipMemcpyHostToDevice);

    // CPU image frames
    cv::Mat in_frame;
    status = video_infile.retrieve(in_frame, 0);
    cv::Mat out_frame(in_frame.size[0], in_frame.size[1], in_frame.type());
    std::cout << "Frame 0 cv_mat_type = " << in_frame.type() << std::endl;

    // GPU image frames
    uchar3 *gpu_in_frame;
    uchar3 *gpu_out_frame;
    int const num_pixels = video_properties.frame_size.width * video_properties.frame_size.height;
    int const num_bytes_per_frame = num_pixels * 3;
    std::cout << "num_pixels = " << num_pixels << std::endl;
    hipError_t cuda_status = hipMalloc((void**) &gpu_in_frame, num_bytes_per_frame);
    assert(cuda_status == hipSuccess && "Unable to allocate input frame on GPU");
    cuda_status = hipMalloc((void**) &gpu_out_frame, num_bytes_per_frame);
    assert(cuda_status == hipSuccess && "Unable to allocate output frame on GPU");

    // Process frames
    for(int i_frame = 0; i_frame < video_properties.num_frames; ++i_frame) {
        std::cout << "Frame " << i_frame << std::endl;

        status = video_infile.read(in_frame);
        assert(status && "Unable to read input frame from video");

        assert(in_frame.isContinuous());
        
        cuda_status = hipMemcpy((void*) gpu_in_frame, (void*) in_frame.data, num_bytes_per_frame, hipMemcpyHostToDevice);
        assert(cuda_status == hipSuccess && "Unable to copy frame into GPU");

        cudaFilter<<<(num_pixels + 1023)/1024, 1024>>>(gpu_out_frame, gpu_in_frame, num_pixels,
                                                        video_properties.frame_size.width, video_properties.frame_size.height,
                                                        filter_cpu.width(), filter_cpu.height());
        cuda_status = hipGetLastError();
        cudaCheckSuccess(cuda_status, "Error launching kernel");
        
        cuda_status = hipMemcpy((void*) out_frame.data, (void*) gpu_out_frame, num_bytes_per_frame, hipMemcpyDeviceToHost);
        cudaCheckSuccess(cuda_status, "Unable to copy frame out of GPU");
        
        video_outfile.write(out_frame);
    }

    return 0;
}